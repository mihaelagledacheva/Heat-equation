
#include <hip/hip_runtime.h>
#include <cassert>
#include <chrono>
#include <functional>
#include <iostream>
#include <vector>

class Point {
public:
    double x, y;
    double U;
    double time;

    Point() {}
    Point(double x, double y, double U=0, double time=0) : x(x), y(y), U(U), time(time) {}

    bool operator==(const Point& other) const {
        return (x == other.x && y == other.y);
    }
};

class Region {
private:
    bool discrete;
    std::function<bool(const Point&)> condition;

public:
    std::vector<Point> points;

    Region() {}
    Region(std::vector<Point> points) : points(points), discrete(true) {}
    Region(std::function<bool(const Point&)> f) : condition(f), discrete(false) {}

    bool contains(const Point& p) const {
        if (discrete) {
            for (const auto& point : points) {
                if (p == point) {
                    return true;
                }
            }
            return false;
        } else {
            return condition(p);
        }
    }
};

//----------------------------------------------------

/**
 * @brief Computes the solutions of the heat equation sequentually
 * @param U_0 - array representing the initial state of the heat function
 * @param U_res - array representing the final state of the heat function
 * @param lambda - the lambda parameter used in the heat equation
 * @param rows - number of rows in the heat function grid
 * @param cols - number of columns in the heat function grid
*/
void ComputeSequential(double* U_0, double* U_res, double lambda, int rows, int cols) {
    // #TODO
}

//----------------------------------------------------

/**
 * @brief Computes the solutions of the heat equation in parallel
 * @param U_0 - array representing the initial state of the heat function
 * @param U_res - array representing the final state of the heat function
 * @param lambda - the lambda parameter used in the heat equation
*/
__global__ void ComputeGPUAux(double* U_0, double* U_res, double lambda) {
    // #TODO
}

/**
 * @brief Parallelizes the numerical scheme for GPU acceleration
 * @param U_0 - array representing the initial state of the heat function
 * @param U_res - array representing the final state of the heat function
 * @param lambda - the lambda parameter used in the heat equation
 * @param rows - number of rows in the heat function grid
 * @param cols - number of columns in the heat function grid
*/
void ComputeGPU(double* U_0, double* U_res, double lambda, int rows, int cols) {
    // #TODO
}

//----------------------------------------------------

/**
 * @brief Computes the solutions of the heat equation in parallel
 * @param U_0 - array representing the initial state of the heat function
 * @param U_res - array representing the final state of the heat function
 * @param lambda - the lambda parameter used in the heat equation
*/
__global__ void ComputeGPUAux2(double* U_0, double* U_res, double lambda) {
    // #TODO
}

/**
 * @brief Parallelizes the numerical scheme for GPU acceleration
 * @param U_0 - array representing the initial state of the heat function
 * @param U_res - array representing the final state of the heat function
 * @param lambda - the lambda parameter used in the heat equation
 * @param rows - number of rows in the heat function grid
 * @param cols - number of columns in the heat function grid
*/
void ComputeGPU2(double* U_0, double* U_res, double lambda, int rows, int cols) {
    // #TODO
}

//----------------------------------------------------

/**
 * @brief Sets values to the parameters of the heat equation
 * @param base - known initial heat values
 * @param target - point at which to evaluate the heat function
 * @param gamma - time step
 * @param delta - space step
*/
void setup(Region& base, Point& target, double& gamma, double& delta) {
    // #TODO
}

/**
 * @brief Fills in the initial heat function grid
 * @param base - known initial heat values
 * @param U_0 - array representing the initial state of the heat function
 * @param gamma - time step
 * @param delta - space step
*/
void initialize(Region& base, double* U_0, double& gamma, double& delta) {
    // #TODO
}

/**
 * @brief Validates the obtained solution
 * @param U_res - array representing the heat function
 * @param i - column at which the target point is located
 * @param j - row at which the target point is located
 * @param res - expected result
*/
void validate(double *U_res, int i, int j, double res) {
    // #TODO
}

/**
 * @brief Visualizes the heat function using gnuplot
 * @param U_res - array representing the heat function
*/
void visualize(double *U_res) {
    // #TODO
}

//----------------------------------------------------

int main() {
    // Parameters
    Region base;
    Point target;
    double gamma;
    double delta;

    setup(base, target, gamma, delta);

    double lambda = gamma / (delta * delta);
    assert(lambda < 0.5);

    // Coordinates of the target point in the heat function grid
    int i = static_cast<int>(target.x/delta);
    int j = static_cast<int>(target.y/delta);
    
    // Dimensions of the problem
    int rows = 2 * j + 1;
    int cols = 2 * i + 1;
    int iterations = static_cast<int>(target.time/gamma);

    // Memory allocation
    double* U_0 = (double*) malloc(rows * cols * sizeof(double));
    double* U_res_seq   = (double*) malloc(rows * cols * sizeof(double));
    double* U_res_cuda  = (double*) malloc(rows * cols * sizeof(double));
    double* U_res_cuda2 = (double*) malloc(rows * cols * sizeof(double));

    initialize(base, U_0, gamma, delta);

    // Sequential algorithm
    auto start_seq = std::chrono::steady_clock::now();
    ComputeSequential(U_0, U_res_seq, lambda, rows, cols);
    auto finish_seq = std::chrono::steady_clock::now();
    auto elapsed_seq = std::chrono::duration_cast<std::chrono::microseconds>(finish_seq - start_seq).count(); 
    std::cout << "Elapsed time for the sequential algorithm: " << elapsed_seq << std::endl << std::endl;
    validate(U_res_seq, i, j, target.U);
    visualize(U_res_seq);

    // GPU algorithm
    auto start_cuda = std::chrono::steady_clock::now();
    ComputeGPU(U_0, U_res_cuda, lambda, rows, cols);
    auto finish_cuda = std::chrono::steady_clock::now();
    auto elapsed_cuda = std::chrono::duration_cast<std::chrono::microseconds>(finish_cuda - start_cuda).count(); 
    std::cout << "Elapsed time for the GPU algorithm: " << elapsed_cuda << std::endl;
    validate(U_res_cuda, i, j, target.U);
    visualize(U_res_cuda);

    // Second GPU algorithm
    auto start_cuda2 = std::chrono::steady_clock::now();
    ComputeGPU2(U_0, U_res_cuda2, lambda, rows, cols);
    auto finish_cuda2 = std::chrono::steady_clock::now();
    auto elapsed_cuda2 = std::chrono::duration_cast<std::chrono::microseconds>(finish_cuda2 - start_cuda2).count(); 
    std::cout << "Elapsed time for the second GPU algorithm: " << elapsed_cuda2 << std::endl;
    validate(U_res_cuda2, i, j, target.U);
    visualize(U_res_cuda2);

    delete[] U_0;
    delete[] U_res_seq;
    delete[] U_res_cuda;
    delete[] U_res_cuda2;

    return 0;
}
