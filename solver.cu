
#include <hip/hip_runtime.h>
/**
 * @brief Computes the solutions of the heat equation sequentually
 * @param U - array representing the heat function
 * @param lambda - the lambda parameter used in the heat equation
 * @param rows - number of rows in the heat function grid
 * @param cols - number of columns in the heat function grid
 * @param iterations - number of iterations
*/
void ComputeSequential1(double* U, double lambda, int rows, int cols, int iterations) {
    double* U_next = (double*) malloc(rows * cols * sizeof(double));
    for (int n = 0; n < iterations; ++n) {
        for (int i = 0; i < rows; ++i) {
            for (int j = 0; j < cols; ++j) {
                double a = (i < rows - 1) ? U[(i + 1) * cols + j] : 0;
                double b = (i > 0) ? U[(i - 1) * cols + j] : 0;
                double c = (j < cols - 1) ? U[i * cols + (j + 1)] : 0;
                double d = (j > 0) ? U[i * cols + (j - 1)] : 0;
                U_next[i * cols + j] = (1 - 4 * lambda) * U[i * cols + j] + lambda * (a + b + c + d);
            }
        }
        std::swap(U, U_next);
    }
    delete[] U_next;
}

/**
 * @brief Computes the solutions of the heat equation in parallel
 * @param U - array representing the current state of the heat function
 * @param U_next - array representing the next state of the heat function
 * @param rows - number of rows in the heat function grid
 * @param cols - number of columns in the heat function grid
 * @param lambda - the lambda parameter used in the heat equation
 * @param size - chunk size
*/
__global__
void ComputeGPUAux1(double* U, double* U_next, int rows, int cols, double lambda, int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int begin = size * index;
    int end = (size * (index + 1) < rows * cols) ? size * (index + 1) : rows * cols;
    
    for (int k = begin; k < end; ++k) {
        int i = k / cols;
        int j = k % cols;

        double a = (i < rows - 1) ? U[(i + 1) * cols + j] : 0;
        double b = (i > 0) ? U[(i - 1) * cols + j] : 0;
        double c = (j < cols - 1) ? U[i * cols + (j + 1)] : 0;
        double d = (j > 0) ? U[i * cols + (j - 1)] : 0;
                
        U_next[i * cols + j] = (1 - 4 * lambda) * U[i * cols + j] + lambda * (a + b + c + d);
    }
}

/**
 * @brief Parallelizes the numerical scheme for GPU acceleration
 * @param U - array representing the state of the heat function
 * @param lambda - the lambda parameter used in the heat equation
 * @param rows - number of rows in the heat function grid
 * @param cols - number of columns in the heat function grid
 * @param iterations - number of iterations
*/
void ComputeGPU1(double* U, double lambda, int rows, int cols, int iterations) {
    const int BLOCKS_NUM = 48;
    const int THREADS_PER_BLOCK = 256;
    const int TOTAL_THREADS = BLOCKS_NUM  * THREADS_PER_BLOCK;
    int size = (rows * cols + TOTAL_THREADS + 1) / TOTAL_THREADS;

    double *d_U, *d_U_next;

    hipMalloc(&d_U, rows * cols * sizeof(double));
    hipMalloc(&d_U_next, rows * cols * sizeof(double));

    hipMemcpy(d_U, U, rows * cols * sizeof(double), hipMemcpyHostToDevice);

    for (int n = 0; n < iterations; ++n) {
        ComputeGPUAux1<<<BLOCKS_NUM, THREADS_PER_BLOCK>>>(d_U, d_U_next, rows, cols, lambda, size);
        hipDeviceSynchronize();
        std::swap(d_U, d_U_next);
    }

    hipMemcpy(U, d_U, rows * cols * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_U);
    hipFree(d_U_next);
}

//----------------------------------------------------
